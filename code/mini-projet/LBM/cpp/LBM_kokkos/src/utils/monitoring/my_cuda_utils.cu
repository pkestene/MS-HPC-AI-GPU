// collection of useful routines

#include "my_cuda_utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// ==================================
// ==================================
/* 
 * initialises CUDA and directs all
 * computations to the given
 * CUDA device
 */
void initCuda(const int selectedDevice)
{

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
  {
    fprintf(stderr, "Sorry, no CUDA device fount");
    exit(1);
  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    exit(2);
  }
  hipSetDevice(selectedDevice);
  checkErrors("initCuda");

  cublasInit();

} // initCuda

// ==================================
// error checking 
// ==================================
void checkErrors(const char *label)
{

  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

} // checkErrors
