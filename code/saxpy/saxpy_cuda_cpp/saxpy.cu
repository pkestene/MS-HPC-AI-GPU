#include "hip/hip_runtime.h"
/**
 * Compute saxpy
 * - on CPU : serial and OpenMP version
 * - on GPU : first using CUDA, then library CuBLAS
 *
 * compare timings.
 *
 */

// =========================
// standard imports
// =========================
#include <stdio.h>
#include <stdlib.h>

// =========================
// CUDA imports
// =========================
#include <hip/hip_runtime.h>
#include <hipblas.h>

// =========================
// OpenMP imports
// =========================
#ifdef _OPENMP
#include <omp.h>
#endif

// =========================
// our imports
// =========================
#include "my_cuda_utils.h"
#include "SimpleTimer.h"
#include "OpenMPTimer.h"
#include "CudaTimer.h"

// =========================
// global variables and configuration section
// =========================

// number of repetitions of the timing loop
// (CPU timers only have a ~ms resolution)
static int numTimingReps = 100;


// =========================
// kernel function (CPU) - serial
// =========================
void saxpy_serial(int n, float alpha, const float *x, float *y)
{

  for (size_t i=0; i<n; i++)
    y[i] = alpha*x[i] + y[i];
}

// =========================
// kernel function (CPU) - OpenMP
// =========================
void saxpy_openmp(int n, float alpha,
                  const float * x, float * y)
{

  #pragma omp parallel for
  #pragma ivdep
  for (size_t i=0; i<n; i++)
    y[i] = alpha*x[i] + y[i];
}


// =========================
// kernel function (CUDA device)
// =========================
__global__ void saxpy_cuda(int n, float alpha, const float *x, float *y)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  // except for special cases, the total number of threads in all blocks
  // adds up to more than the vector length n, so this conditional is
  // EXTREMELY important to avoid writing past the allocated memory for
  // the vector y.
  if (i<n)
    y[i] = alpha*x[i] + y[i];
}


// =========================
// main routine
// =========================
int main (int argc, char **argv)
{

  // base-2 log of array size
  int log2N = argc > 1 ? atoi(argv[1]) : 22;

  // problem size (vector length) N
  //size_t N = 1234567;
  size_t N = 1 << log2N;
  //size_t N = 40000;

  SimpleTimer cpuTimer;
  OpenMPTimer ompTimer;
  CudaTimer   gpuTimer;

#ifdef _OPENMP
  int threadId;
#pragma omp parallel private(threadId)
  {
    threadId = omp_get_thread_num();
    if (threadId==0)
      printf("Using %d OpenMP threads\n",omp_get_num_threads());
  }
#else
  printf("OpenMP not activated\n");
#endif

  // =========================
  // (1) initialisations:
  //     implemented in my_cuda_utils.c
  // =========================
  initCuda(0);


  // =========================
  // (2) allocate memory on host (main CPU memory) and device,
  //     h_ denotes data residing on the host, d_ on device
  // =========================
  float *h_x = (float*)malloc(N*sizeof(float));
  float *h_y = (float*)malloc(N*sizeof(float));
  float *d_x;
  hipMalloc((void**)&d_x, N*sizeof(float));
  float *d_y;
  hipMalloc((void**)&d_y, N*sizeof(float));
  checkErrors("memory allocation");


  // =========================
  // (3) initialise data on the CPU
  // =========================
//#pragma omp parallel for
  for (size_t i=0; i<N; i++)
  {
    h_x[i] = 1.0f + i;
    h_y[i] = (float)(N-i+1);
  }


  // =========================
  // (4) copy data to device
  // =========================
  hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);
  checkErrors("copy data to device");


  // =========================
  // (5a) perform computation on host - SERIAL
  //     use our straight forward code
  //     and our utility functions to time everything,
  //     note that gettimeofday has ~ms resolution, so
  //     perform everything in a loop to minimise
  //     timing noise
  // =========================
  float alpha = 2.0;
  cpuTimer.start();
  for (int iter=0; iter<numTimingReps; iter++)
    saxpy_serial(N, alpha, h_x, h_y);
  cpuTimer.stop();
  double elapsed = cpuTimer.elapsed();
  printf("CPU CODE (Serial): %8ld elements, %10.6f ms per iteration, %6.3f GFLOP/s, %7.3f GB/s\n",
         N,
         (elapsed*1000.0)/(double)numTimingReps,
         2.0*N*numTimingReps / (elapsed*1e9),
         3.0*N*sizeof(float)*numTimingReps / (elapsed*1e9) );

  // =========================
  // (5b) perform computation on host - OpenMP
  // =========================
  ompTimer.start();
  for (int iter=0; iter<numTimingReps; iter++)
    saxpy_openmp(N, alpha, h_x, h_y);
  ompTimer.stop();
  elapsed = ompTimer.elapsed();
  printf("CPU CODE (OpenMP): %8ld elements, %10.6f ms per iteration, %6.3f GFLOP/s, %7.3f GB/s\n",
         N,
         (elapsed*1000.0)/(double)numTimingReps,
         2.0*N*numTimingReps / (elapsed*1e9),
         3.0*N*sizeof(float)*numTimingReps / (elapsed*1e9) );


  // =========================
  // (7) perform computation on device, our implementation
  //     use CUDA events to time the execution:
  //     (a) insert "tag" into instruction stream
  //     (b) execute kernel
  //     (c) insert another tag into instruction stream
  //     (d) synchronize (ie, wait for) this tag (event)
  //     CUDA events have a resolution of ~0.5us
  // =========================
  float time;

  // Mapping onto the device:
  // - each thread computes one element of the output array in situ
  // - all threads and blocks are independent
  // - use 256 threads per block
  // - use as many blocks as necessary (the last block is not entirely
  //   full if n is not a multiple of 256)
  int numThreadsPerBlock = 128;
  int numBlocks = (N+numThreadsPerBlock-1) / numThreadsPerBlock;

  gpuTimer.start();
  saxpy_cuda<<<numBlocks, numThreadsPerBlock>>>(N, alpha, d_x, d_y);
  gpuTimer.stop();
  time = gpuTimer.elapsed();
  printf("GPU CODE (CUDA)  : %8ld elements, %10.6f ms per iteration, %6.3f GFLOP/s, %7.3f GB/s\n",
         N,
         time*1000,
         2.0*N / (time*1e9),
         3.0*N*sizeof(float) / (time*1e9) );


  // =========================
  // (8) read back result from device into temp vector
  // =========================
  float *h_z = (float*)malloc(N*sizeof(float));
  hipMemcpy(h_z, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");


  // =========================
  // (9) perform computation on device, CUBLAS
  // =========================
  {
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);

	gpuTimer.reset();
	gpuTimer.start();
	hipblasSaxpy(handle, N, &alpha, d_x, 1, d_y, 1);
	gpuTimer.stop();
	time = gpuTimer.elapsed();
	printf("GPU CODE (CUBLAS): %8ld elements, %10.6f ms per iteration, %6.3f GFLOP/s, %7.3f GB/s\n",
		   N,
		   time*1000,
		   2.0*N / (time*1e9),
		   3.0*N*sizeof(float) / (time*1e9) );
	status = hipblasDestroy(handle);
  }

  // =========================
  // (10) perform result comparison
  //      we need to re-run the CPU code because
  //      it has been executed 1000 times before
  // =========================
  int errorCount = 0;
  for (size_t i=0; i<N; i++)
  {
    h_x[i] = 1.0f + i;
    h_y[i] = (float)(N-i+1);
  }
  saxpy_serial(N, alpha, h_x, h_y);
  for (size_t i=0; i<N; i++)
  {
    if (abs(h_y[i]-h_z[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");



  // =========================
  // (11) clean up, free memory
  // =========================
  free(h_x);
  free(h_y);
  free(h_z);
  hipFree(d_x);
  hipFree(d_y);

  return 0;
}
